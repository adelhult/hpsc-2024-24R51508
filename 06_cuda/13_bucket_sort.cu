
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>

// I'm using the same approach as I did for the Open MPI homework. That is, 
// 1. Count the frequency (fill the buckets)
// 2. Use the provided prefix sum on the buckets to calculate the offsets 
// 3. Remove bucket[i] from each offset since we want an exclusive prefix sum
//    Note: a *much* nicer approach would of course be to use an exclusive scan from the start
//          for instance this one found in Nvida's 'GPU Gems 3': 
//          https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
//          but I want to keep things simple so I understand what I'm doing :)
// 4. Finally, go through the list and sort the list in parallel using the offsets

// Fill the buckets by counting the frequency of each value in the array 'xs'
__global__ void frequency(int *keys, int *buckets, int len) {
  const auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= len) {
    return; // we have enough threads already
  }

  atomicAdd(&buckets[keys[i]], 1);
}

// Prefix sum taken from the '08_scan.cu' example
__global__ void prefix_sum(int *a, int *b, int len) {
  const auto i = blockIdx.x * blockDim.x + threadIdx.x;
  for (int j=1; j < len; j<<=1) {
    b[i] = a[i];
    __syncthreads();
    if (i>=j) {
      a[i] += b[i-j];
    }
    __syncthreads();
  }
}

__global__ void make_exclusive(int *xs, int *ys, int len) {
  const auto i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= len) {
    return; // we have enough threads already
  }

  xs[i] -= ys[i];
}

// Use the offsets to sort the array 'keys'
__global__ void sort(int *keys, int *buckets, int *offsets, int len) {
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= len) {
    return; // we have enough threads already
  }

  // Go through 'keys' and fill it with the correct value using
  // 'j' as a cursor (starting at the correct offset)
  for (auto j = offsets[i]; buckets[i] > 0; buckets[i]--) {
      keys[j++] = i;
  }
}

int main() {
  const int n = 10;
  const int range = 5;
  
  int *keys, *buckets, *offsets;
  hipMallocManaged(&keys,    n*sizeof(int));
  hipMallocManaged(&buckets, range*sizeof(int));
  hipMallocManaged(&offsets, range*sizeof(int));

  // Fill the the 'keys' array with random values within the range
  for (int i=0; i<n; i++) {
    keys[i] = rand() % range;
    printf("%d ",keys[i]);
  }
  printf("\n");

  // Set all elements in the buckets array to zero at the start
  for (int i=0; i<range; i++) {
    buckets[i] = 0;
  }

  hipDeviceSynchronize();
  // Now, let's actualy start the bucket sort

  frequency<<<1,n>>>(keys, buckets, n); // TODO nr threads
  hipDeviceSynchronize();

  int *temp;
  hipMalloc(&temp, range * sizeof(int));
  std::memcpy(offsets, buckets, range * sizeof(int));
  prefix_sum<<<1, range>>>(offsets, temp, range);
  hipDeviceSynchronize();

  make_exclusive<<<1, range>>>(offsets, buckets, range);
  hipDeviceSynchronize();

  sort<<<1, range>>>(keys, buckets, offsets, range);
  hipDeviceSynchronize();

  // Print the sorted array
  for (int i=0; i<n; i++) {
    printf("%d ",keys[i]);
  }
  printf("\n");
}
